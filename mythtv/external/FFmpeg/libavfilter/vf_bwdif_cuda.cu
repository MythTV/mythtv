
#include <hip/hip_runtime.h>
/*
 * Copyright (C) 2019 Philip Langdale <philipl@overt.org>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

__device__ static const int coef_lf[2] = { 4309, 213 };
__device__ static const int coef_hf[3] = { 5570, 3801, 1016 };
__device__ static const int coef_sp[2] = { 5077, 981 };

template<typename T>
__inline__ __device__ T max3(T a, T b, T c)
{
    T x = max(a, b);
    return max(x, c);
}

template<typename T>
__inline__ __device__ T min3(T a, T b, T c)
{
    T x = min(a, b);
    return min(x, c);
}

template<typename T>
__inline__ __device__ T clip(T a, T min, T max)
{
    if (a < min) {
        return min;
    } else if (a > max) {
        return max;
    } else {
        return a;
    }
}

template<typename T>
__inline__ __device__ T filter_intra(T cur_prefs3, T cur_prefs,
                                     T cur_mrefs, T cur_mrefs3,
                                     int clip_max)
{
    int final = (coef_sp[0] * (cur_mrefs + cur_prefs) -
                 coef_sp[1] * (cur_mrefs3 + cur_prefs3)) >> 13;
    return clip(final, 0, clip_max);
}

template<typename T>
__inline__ __device__ T filter(T cur_prefs3, T cur_prefs, T cur_mrefs, T cur_mrefs3,
                               T prev2_prefs4, T prev2_prefs2, T prev2_0, T prev2_mrefs2, T prev2_mrefs4,
                               T prev_prefs, T prev_mrefs, T next_prefs, T next_mrefs,
                               T next2_prefs4, T next2_prefs2, T next2_0, T next2_mrefs2, T next2_mrefs4,
                               int clip_max)
{
    T final;

    int c = cur_mrefs;
    int d = (prev2_0 + next2_0) >> 1;
    int e = cur_prefs;

    int temporal_diff0 = abs(prev2_0 - next2_0);
    int temporal_diff1 = (abs(prev_mrefs - c) + abs(prev_prefs - e)) >> 1;
    int temporal_diff2 = (abs(next_mrefs - c) + abs(next_prefs - e)) >> 1;
    int diff = max3(temporal_diff0 >> 1, temporal_diff1, temporal_diff2);

    if (!diff) {
        final = d;
    } else {
        int b = ((prev2_mrefs2 + next2_mrefs2) >> 1) - c;
        int f = ((prev2_prefs2 + next2_prefs2) >> 1) - e;
        int dc = d - c;
        int de = d - e;
        int mmax = max3(de, dc, min(b, f));
        int mmin = min3(de, dc, max(b, f));
        diff = max3(diff, mmin, -mmax);

        int interpol;
        if (abs(c - e) > temporal_diff0) {
            interpol = (((coef_hf[0] * (prev2_0 + next2_0)
                - coef_hf[1] * (prev2_mrefs2 + next2_mrefs2 + prev2_prefs2 + next2_prefs2)
                + coef_hf[2] * (prev2_mrefs4 + next2_mrefs4 + prev2_prefs4 + next2_mrefs4)) >> 2)
                + coef_lf[0] * (c + e) - coef_lf[1] * (cur_mrefs3 + cur_prefs3)) >> 13;
        } else {
            interpol = (coef_sp[0] * (c + e) - coef_sp[1] * (cur_mrefs3 + cur_prefs3)) >> 13;
        }

        if (interpol > d + diff) {
            interpol = d + diff;
        } else if (interpol < d - diff) {
            interpol = d - diff;
        }
        final = clip(interpol, 0, clip_max);
    }

    return final;
}

template<typename T>
__inline__ __device__ void bwdif_single(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff,
                                        int is_field_end, int clip_max)
{
    // Identify location
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    // Don't modify the primary field
    if (yo % 2 == parity) {
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    T cur_prefs3 = tex2D<T>(cur, xo, yo + 3);
    T cur_prefs = tex2D<T>(cur, xo, yo + 1);
    T cur_mrefs = tex2D<T>(cur, xo, yo - 1);
    T cur_mrefs3 = tex2D<T>(cur, xo, yo - 3);

    if (is_field_end) {
        dst[yo*dst_pitch+xo] =
            filter_intra(cur_prefs3, cur_prefs, cur_mrefs, cur_mrefs3, clip_max);
        return;
    }

    // Calculate temporal prediction
    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T prev2_prefs4 = tex2D<T>(prev2, xo,  yo + 4);
    T prev2_prefs2 = tex2D<T>(prev2, xo,  yo + 2);
    T prev2_0 = tex2D<T>(prev2, xo,  yo + 0);
    T prev2_mrefs2 = tex2D<T>(prev2, xo,  yo - 2);
    T prev2_mrefs4 = tex2D<T>(prev2, xo,  yo - 4);
    T prev_prefs = tex2D<T>(prev1, xo,  yo + 1);
    T prev_mrefs = tex2D<T>(prev1, xo,  yo - 1);
    T next_prefs = tex2D<T>(next1, xo,  yo + 1);
    T next_mrefs = tex2D<T>(next1, xo,  yo - 1);
    T next2_prefs4 = tex2D<T>(next2, xo,  yo + 4);
    T next2_prefs2 = tex2D<T>(next2, xo,  yo + 2);
    T next2_0 = tex2D<T>(next2, xo,  yo + 0);
    T next2_mrefs2 = tex2D<T>(next2, xo,  yo - 2);
    T next2_mrefs4 = tex2D<T>(next2, xo,  yo - 4);

    dst[yo*dst_pitch+xo] = filter(cur_prefs3, cur_prefs, cur_mrefs, cur_mrefs3,
                                  prev2_prefs4, prev2_prefs2, prev2_0, prev2_mrefs2, prev2_mrefs4,
                                  prev_prefs, prev_mrefs, next_prefs, next_mrefs,
                                  next2_prefs4, next2_prefs2, next2_0, next2_mrefs2, next2_mrefs4,
                                  clip_max);
}

template <typename T>
__inline__ __device__ void bwdif_double(T *dst,
                                        hipTextureObject_t prev,
                                        hipTextureObject_t cur,
                                        hipTextureObject_t next,
                                        int dst_width, int dst_height, int dst_pitch,
                                        int src_width, int src_height,
                                        int parity, int tff,
                                        int is_field_end, int clip_max)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (xo >= dst_width || yo >= dst_height) {
        return;
    }

    if (yo % 2 == parity) {
      // Don't modify the primary field
      dst[yo*dst_pitch+xo] = tex2D<T>(cur, xo, yo);
      return;
    }

    T cur_prefs3 = tex2D<T>(cur, xo, yo + 3);
    T cur_prefs = tex2D<T>(cur, xo, yo + 1);
    T cur_mrefs = tex2D<T>(cur, xo, yo - 1);
    T cur_mrefs3 = tex2D<T>(cur, xo, yo - 3);

    if (is_field_end) {
        T final;
        final.x = filter_intra(cur_prefs3.x, cur_prefs.x, cur_mrefs.x, cur_mrefs3.x,
                               clip_max);
        final.y = filter_intra(cur_prefs3.y, cur_prefs.y, cur_mrefs.y, cur_mrefs3.y,
                               clip_max);
        dst[yo*dst_pitch+xo] = final;
        return;
    }

    int is_second_field = !(parity ^ tff);

    hipTextureObject_t prev2 = prev;
    hipTextureObject_t prev1 = is_second_field ? cur : prev;
    hipTextureObject_t next1 = is_second_field ? next : cur;
    hipTextureObject_t next2 = next;

    T prev2_prefs4 = tex2D<T>(prev2, xo,  yo + 4);
    T prev2_prefs2 = tex2D<T>(prev2, xo,  yo + 2);
    T prev2_0 = tex2D<T>(prev2, xo,  yo + 0);
    T prev2_mrefs2 = tex2D<T>(prev2, xo,  yo - 2);
    T prev2_mrefs4 = tex2D<T>(prev2, xo,  yo - 4);
    T prev_prefs = tex2D<T>(prev1, xo,  yo + 1);
    T prev_mrefs = tex2D<T>(prev1, xo,  yo - 1);
    T next_prefs = tex2D<T>(next1, xo,  yo + 1);
    T next_mrefs = tex2D<T>(next1, xo,  yo - 1);
    T next2_prefs4 = tex2D<T>(next2, xo,  yo + 4);
    T next2_prefs2 = tex2D<T>(next2, xo,  yo + 2);
    T next2_0 = tex2D<T>(next2, xo,  yo + 0);
    T next2_mrefs2 = tex2D<T>(next2, xo,  yo - 2);
    T next2_mrefs4 = tex2D<T>(next2, xo,  yo - 4);

    T final;
    final.x = filter(cur_prefs3.x, cur_prefs.x, cur_mrefs.x, cur_mrefs3.x,
                     prev2_prefs4.x, prev2_prefs2.x, prev2_0.x, prev2_mrefs2.x, prev2_mrefs4.x,
                     prev_prefs.x, prev_mrefs.x, next_prefs.x, next_mrefs.x,
                     next2_prefs4.x, next2_prefs2.x, next2_0.x, next2_mrefs2.x, next2_mrefs4.x,
                     clip_max);
    final.y = filter(cur_prefs3.y, cur_prefs.y, cur_mrefs.y, cur_mrefs3.y,
                     prev2_prefs4.y, prev2_prefs2.y, prev2_0.y, prev2_mrefs2.y, prev2_mrefs4.y,
                     prev_prefs.y, prev_mrefs.y, next_prefs.y, next_mrefs.y,
                     next2_prefs4.y, next2_prefs2.y, next2_0.y, next2_mrefs2.y, next2_mrefs4.y,
                     clip_max);

    dst[yo*dst_pitch+xo] = final;
}

extern "C" {

__global__ void bwdif_uchar(unsigned char *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, int is_field_end, int clip_max)
{
    bwdif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, is_field_end, clip_max);
}

__global__ void bwdif_ushort(unsigned short *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, int is_field_end, int clip_max)
{
    bwdif_single(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, is_field_end, clip_max);
}

__global__ void bwdif_uchar2(uchar2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, int is_field_end, int clip_max)
{
    bwdif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, is_field_end, clip_max);
}

__global__ void bwdif_ushort2(ushort2 *dst,
                            hipTextureObject_t prev,
                            hipTextureObject_t cur,
                            hipTextureObject_t next,
                            int dst_width, int dst_height, int dst_pitch,
                            int src_width, int src_height,
                            int parity, int tff, int is_field_end, int clip_max)
{
    bwdif_double(dst, prev, cur, next,
                 dst_width, dst_height, dst_pitch,
                 src_width, src_height,
                 parity, tff, is_field_end, clip_max);
}

} /* extern "C" */
