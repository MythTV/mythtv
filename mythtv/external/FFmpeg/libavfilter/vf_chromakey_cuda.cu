#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022 Mohamed Khaled <Mohamed_Khaled_Kamal@outlook.com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include "cuda/vector_helpers.cuh"

extern "C"
{

/**
 * @brief function contains the main logic of chroma keying, and changes the alpahc channel with the suitable value
 *
 * @param src_tex           texture U or texture UV , decided based on the passed is_uchar2 flag
 * @param src_tex_V         texture V , used only if is_uchar2 flag is false
 * @param dst_A             alpha channel destination
 * @param width_uv          width of uv channels
 * @param height_uv         height of uv channels
 * @param width             width of alpha channel
 * @param height            height of alpha channel
 * @param pitch             pitch of alpha channel
 * @param x                 current x coordinate of pixel
 * @param y                 current y coordinate of pixel
 * @param chromakey_uv      uv values for chroma keying
 * @param similarity        similarity of keying
 * @param blend             blend of keying
 */
__device__ static inline void change_alpha_channel(
    hipTextureObject_t src_tex, hipTextureObject_t src_tex_V, uchar *dst_A,
    int width_uv, int height_uv,
    int width, int height, int pitch,
    int x, int y,
    float2 chromakey_uv, float similarity, float blend)
{
    int window_size = 3;
    int start_r = x - window_size / 2;
    int start_c = y - window_size / 2;
    int resize_ratio = width / width_uv;
    int counter = 0;
    float diff = 0.0f;
    float du, dv;
    uchar alpha_value;

    // loop over the eight neighbourhood of the current pixel(x,y)
    for (uchar i = 0; i < window_size; i++)
    {
        for (uchar j = 0; j < window_size; j++)
        {
            float u_value, v_value;
            int r = start_r + i;
            int c = start_c + j;

            if (r < 0 || r >= width_uv || c < 0 || c >= height_uv)
                continue;

            if (!src_tex_V) {
                float2 temp_uv = tex2D<float2>(src_tex, r, c);
                u_value = temp_uv.x;
                v_value = temp_uv.y;
            } else {
                u_value = tex2D<float>(src_tex, r, c);
                v_value = tex2D<float>(src_tex_V, r, c);
            }

            du = (u_value * 255.0f) - chromakey_uv.x;
            dv = (v_value * 255.0f) - chromakey_uv.y;
            diff += sqrtf((du * du + dv * dv) / (255.0f * 255.0f * 2.f));

            counter++;
        }
    }

    if (counter > 0)
        diff = diff / counter;
    else
        diff /= 9.0f;

    if (blend>0.0001f)
        alpha_value = __saturatef((diff - similarity) / blend) * 255;
    else
        alpha_value = (diff < similarity) ? 0 : 255;

    //write the value in the alpha channel with regarding the ratio of (alpha_size : uv_size)
    for (uchar k = 0; k < resize_ratio; k++)
    {
        for (uchar l = 0; l < resize_ratio; l++)
        {
            int x_resize = x * resize_ratio + k;
            int y_resize = y * resize_ratio + l;
            int a_channel_resize = y_resize * pitch + x_resize;

            if (y_resize >= height || x_resize >= width)
                continue;

            dst_A[a_channel_resize] = alpha_value;
        }
    }
}

__global__ void Process_uchar(
    hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_U, hipTextureObject_t src_tex_V,
    uchar *dst_Y, uchar *dst_U, uchar *dst_V, uchar *dst_A,
    int width, int height, int pitch,
    int width_uv, int height_uv, int pitch_uv,
    float u_key, float v_key, float similarity, float blend)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;

    dst_Y[y * pitch + x] = tex2D<float>(src_tex_Y, x, y)*255;

    if (y >= height_uv || x >= width_uv)
        return;

    int uv_index = y * pitch_uv + x;
    dst_U[uv_index] = tex2D<float>(src_tex_U, x, y) * 255;
    dst_V[uv_index] = tex2D<float>(src_tex_V, x, y) * 255;

    change_alpha_channel(src_tex_U, src_tex_V, dst_A,
                         width_uv, height_uv,
                         width, height, pitch,
                         x, y,
                         make_float2(u_key, v_key), similarity, blend);
}

__global__ void Process_uchar2(
    hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_UV, hipTextureObject_t unused1,
    uchar *dst_Y, uchar *dst_U, uchar *dst_V, uchar *dst_A,
    int width, int height, int pitch,
    int width_uv, int height_uv,int pitch_uv,
    float u_key, float v_key, float similarity, float blend)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // x coordinate of current pixel
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // y coordinate of current pixel

    if (y >= height || x >= width)
        return;

    dst_Y[y * pitch + x] = tex2D<float>(src_tex_Y, x, y) * 255;

    if (y >= height_uv || x >= width_uv)
        return;

    int uv_index = y * pitch_uv + x;
    float2 uv_temp = tex2D<float2>(src_tex_UV, x, y);
    dst_U[uv_index] = uv_temp.x * 255;
    dst_V[uv_index] = uv_temp.y * 255;

    change_alpha_channel(src_tex_UV, (hipTextureObject_t)nullptr,
                         dst_A, width_uv, height_uv,
                         width, height, pitch,
                         x, y,
                         make_float2(u_key, v_key), similarity, blend);
}

}
