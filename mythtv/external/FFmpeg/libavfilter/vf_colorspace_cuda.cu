
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {
#define MPEG_LUMA_MIN   (16)
#define MPEG_CHROMA_MIN (16)
#define MPEG_LUMA_MAX   (235)
#define MPEG_CHROMA_MAX (240)

#define JPEG_LUMA_MIN   (0)
#define JPEG_CHROMA_MIN (1)
#define JPEG_LUMA_MAX   (255)
#define JPEG_CHROMA_MAX (255)

__device__ int mpeg_min[] = {MPEG_LUMA_MIN, MPEG_CHROMA_MIN};
__device__ int mpeg_max[] = {MPEG_LUMA_MAX, MPEG_CHROMA_MAX};

__device__ int jpeg_min[] = {JPEG_LUMA_MIN, JPEG_CHROMA_MIN};
__device__ int jpeg_max[] = {JPEG_LUMA_MAX, JPEG_CHROMA_MAX};

__device__ int clamp(int val, int min, int max)
{
    if (val < min)
        return min;
    else if (val > max)
        return max;
    else
        return val;
}

__global__ void to_jpeg_cuda(const unsigned char* src, unsigned char* dst,
                             int pitch, int comp_id)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int src_, dst_;

    // 8 bit -> 15 bit for better precision
    src_ = static_cast<int>(src[x + y * pitch]) << 7;

    // Conversion
    dst_ = comp_id ? (min(src_, 30775) * 4663 - 9289992) >> 12    // chroma
                   : (min(src_, 30189) * 19077 - 39057361) >> 14; // luma

    // Dither replacement
    dst_ = dst_ + 64;

    // Back to 8 bit
    dst_ = clamp(dst_ >> 7, jpeg_min[comp_id], jpeg_max[comp_id]);
    dst[x + y * pitch] = static_cast<unsigned char>(dst_);
}

__global__ void to_mpeg_cuda(const unsigned char* src, unsigned char* dst,
                             int pitch, int comp_id)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int src_, dst_;

    // 8 bit -> 15 bit for better precision
    src_ = static_cast<int>(src[x + y * pitch]) << 7;

    // Conversion
    dst_ = comp_id ? (src_ * 1799 + 4081085) >> 11    // chroma
                   : (src_ * 14071 + 33561947) >> 14; // luma

    // Dither replacement
    dst_ = dst_ + 64;

    // Back to 8 bit
    dst_ = clamp(dst_ >> 7, mpeg_min[comp_id], mpeg_max[comp_id]);
    dst[x + y * pitch] = static_cast<unsigned char>(dst_);
}

}
