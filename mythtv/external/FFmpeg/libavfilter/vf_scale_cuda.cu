#include "hip/hip_runtime.h"
/*
 * This file is part of FFmpeg.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"
#include "vf_scale_cuda.h"

template<typename T>
using subsample_function_t = T (*)(hipTextureObject_t tex, int xo, int yo,
                                   int dst_width, int dst_height,
                                   int src_width, int src_height,
                                   int bit_depth, float param);

// --- CONVERSION LOGIC ---

static const ushort mask_10bit = 0xFFC0;
static const ushort mask_16bit = 0xFFFF;

static inline __device__ ushort conv_8to16(uchar in, ushort mask)
{
    return ((ushort)in | ((ushort)in << 8)) & mask;
}

static inline __device__ uchar conv_16to8(ushort in)
{
    return in >> 8;
}

static inline __device__ uchar conv_10to8(ushort in)
{
    return in >> 8;
}

static inline __device__ ushort conv_10to16(ushort in)
{
    return in | (in >> 10);
}

static inline __device__ ushort conv_16to10(ushort in)
{
    return in & mask_10bit;
}

#define DEF_F(N, T) \
    template<subsample_function_t<in_T> subsample_func_y,                                      \
             subsample_function_t<in_T_uv> subsample_func_uv>                                  \
    __device__ static inline void N(hipTextureObject_t src_tex[4], T *dst[4], int xo, int yo, \
                                    int dst_width, int dst_height, int dst_pitch,              \
                                    int src_width, int src_height, float param)

#define SUB_F(m, plane) \
    subsample_func_##m(src_tex[plane], xo, yo, \
                       dst_width, dst_height,  \
                       src_width, src_height,  \
                       in_bit_depth, param)

// FFmpeg passes pitch in bytes, CUDA uses potentially larger types
#define FIXED_PITCH \
    (dst_pitch/sizeof(*dst[0]))

#define DEFAULT_DST(n) \
    dst[n][yo*FIXED_PITCH+xo]

// yuv420p->X

struct Convert_yuv420p_yuv420p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
        DEFAULT_DST(2) = SUB_F(uv, 2);
    }
};

struct Convert_yuv420p_nv12
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_uchar2(
            SUB_F(uv, 1),
            SUB_F(uv, 2)
        );
    }
};

struct Convert_yuv420p_yuv444p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
        DEFAULT_DST(2) = SUB_F(uv, 2);
    }
};

struct Convert_yuv420p_p010le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_10bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(SUB_F(uv, 1), mask_10bit),
            conv_8to16(SUB_F(uv, 2), mask_10bit)
        );
    }
};

struct Convert_yuv420p_p016le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(SUB_F(uv, 1), mask_16bit),
            conv_8to16(SUB_F(uv, 2), mask_16bit)
        );
    }
};

struct Convert_yuv420p_yuv444p16le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = conv_8to16(SUB_F(uv, 1), mask_16bit);
        DEFAULT_DST(2) = conv_8to16(SUB_F(uv, 2), mask_16bit);
    }
};

// nv12->X

struct Convert_nv12_yuv420p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = res.x;
        DEFAULT_DST(2) = res.y;
    }
};

struct Convert_nv12_nv12
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
    }
};

struct Convert_nv12_yuv444p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = res.x;
        DEFAULT_DST(2) = res.y;
    }
};

struct Convert_nv12_p010le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_10bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(res.x, mask_10bit),
            conv_8to16(res.y, mask_10bit)
        );
    }
};

struct Convert_nv12_p016le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(res.x, mask_16bit),
            conv_8to16(res.y, mask_16bit)
        );
    }
};

struct Convert_nv12_yuv444p16le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar2 in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_8to16(res.x, mask_16bit);
        DEFAULT_DST(2) = conv_8to16(res.y, mask_16bit);
    }
};

// yuv444p->X

struct Convert_yuv444p_yuv420p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
        DEFAULT_DST(2) = SUB_F(uv, 2);
    }
};

struct Convert_yuv444p_nv12
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_uchar2(
            SUB_F(uv, 1),
            SUB_F(uv, 2)
        );
    }
};

struct Convert_yuv444p_yuv444p
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
        DEFAULT_DST(2) = SUB_F(uv, 2);
    }
};

struct Convert_yuv444p_p010le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_10bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(SUB_F(uv, 1), mask_10bit),
            conv_8to16(SUB_F(uv, 2), mask_10bit)
        );
    }
};

struct Convert_yuv444p_p016le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            conv_8to16(SUB_F(uv, 1), mask_16bit),
            conv_8to16(SUB_F(uv, 2), mask_16bit)
        );
    }
};

struct Convert_yuv444p_yuv444p16le
{
    static const int in_bit_depth = 8;
    typedef uchar in_T;
    typedef uchar in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_8to16(SUB_F(y, 0), mask_16bit);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = conv_8to16(SUB_F(uv, 1), mask_16bit);
        DEFAULT_DST(2) = conv_8to16(SUB_F(uv, 2), mask_16bit);
    }
};

// p010le->X

struct Convert_p010le_yuv420p
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_10to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_10to8(res.x);
        DEFAULT_DST(2) = conv_10to8(res.y);
    }
};

struct Convert_p010le_nv12
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_10to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_uchar2(
            conv_10to8(res.x),
            conv_10to8(res.y)
        );
    }
};

struct Convert_p010le_yuv444p
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_10to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_10to8(res.x);
        DEFAULT_DST(2) = conv_10to8(res.y);
    }
};

struct Convert_p010le_p010le
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
    }
};

struct Convert_p010le_p016le
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_10to16(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_ushort2(
            conv_10to16(res.x),
            conv_10to16(res.y)
        );
    }
};

struct Convert_p010le_yuv444p16le
{
    static const int in_bit_depth = 10;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_10to16(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_10to16(res.x);
        DEFAULT_DST(2) = conv_10to16(res.y);
    }
};

// p016le->X

struct Convert_p016le_yuv420p
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_16to8(res.x);
        DEFAULT_DST(2) = conv_16to8(res.y);
    }
};

struct Convert_p016le_nv12
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_uchar2(
            conv_16to8(res.x),
            conv_16to8(res.y)
        );
    }
};

struct Convert_p016le_yuv444p
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = conv_16to8(res.x);
        DEFAULT_DST(2) = conv_16to8(res.y);
    }
};

struct Convert_p016le_p010le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to10(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = make_ushort2(
            conv_16to10(res.x),
            conv_16to10(res.y)
        );
    }
};

struct Convert_p016le_p016le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
    }
};

struct Convert_p016le_yuv444p16le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort2 in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        in_T_uv res = SUB_F(uv, 1);
        DEFAULT_DST(1) = res.x;
        DEFAULT_DST(2) = res.y;
    }
};

// yuv444p16le->X

struct Convert_yuv444p16le_yuv420p
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = conv_16to8(SUB_F(uv, 1));
        DEFAULT_DST(2) = conv_16to8(SUB_F(uv, 2));
    }
};

struct Convert_yuv444p16le_nv12
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef uchar out_T;
    typedef uchar2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_uchar2(
            conv_16to8(SUB_F(uv, 1)),
            conv_16to8(SUB_F(uv, 2))
        );
    }
};

struct Convert_yuv444p16le_yuv444p
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef uchar out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to8(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = conv_16to8(SUB_F(uv, 1));
        DEFAULT_DST(2) = conv_16to8(SUB_F(uv, 2));
    }
};

struct Convert_yuv444p16le_p010le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = conv_16to10(SUB_F(y, 0));
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            conv_16to10(SUB_F(uv, 1)),
            conv_16to10(SUB_F(uv, 2))
        );
    }
};

struct Convert_yuv444p16le_p016le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef ushort out_T;
    typedef ushort2 out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = make_ushort2(
            SUB_F(uv, 1),
            SUB_F(uv, 2)
        );
    }
};

struct Convert_yuv444p16le_yuv444p16le
{
    static const int in_bit_depth = 16;
    typedef ushort in_T;
    typedef ushort in_T_uv;
    typedef ushort out_T;
    typedef ushort out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
        DEFAULT_DST(1) = SUB_F(uv, 1);
        DEFAULT_DST(2) = SUB_F(uv, 2);
    }
};

// bgr0->X

struct Convert_bgr0_bgr0
{
    static const int in_bit_depth = 8;
    typedef uchar4 in_T;
    typedef uchar in_T_uv;
    typedef uchar4 out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
    }
};

struct Convert_bgr0_rgb0
{
    static const int in_bit_depth = 8;
    typedef uchar4 in_T;
    typedef uchar in_T_uv;
    typedef uchar4 out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        uchar4 res = SUB_F(y, 0);
        DEFAULT_DST(0) = make_uchar4(
            res.z,
            res.y,
            res.x,
            res.w
        );
    }

    DEF_F(Convert_uv, out_T_uv)
    {
    }
};

// rgb0->X

struct Convert_rgb0_bgr0
{
    static const int in_bit_depth = 8;
    typedef uchar4 in_T;
    typedef uchar in_T_uv;
    typedef uchar4 out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        uchar4 res = SUB_F(y, 0);
        DEFAULT_DST(0) = make_uchar4(
            res.z,
            res.y,
            res.x,
            res.w
        );
    }

    DEF_F(Convert_uv, out_T_uv)
    {
    }
};

struct Convert_rgb0_rgb0
{
    static const int in_bit_depth = 8;
    typedef uchar4 in_T;
    typedef uchar in_T_uv;
    typedef uchar4 out_T;
    typedef uchar out_T_uv;

    DEF_F(Convert, out_T)
    {
        DEFAULT_DST(0) = SUB_F(y, 0);
    }

    DEF_F(Convert_uv, out_T_uv)
    {
    }
};

// --- SCALING LOGIC ---

typedef float4 (*coeffs_function_t)(float, float);

__device__ static inline float4 lanczos_coeffs(float x, float param)
{
    const float pi = 3.141592654f;

    float4 res = make_float4(
        pi * (x + 1),
        pi * x,
        pi * (x - 1),
        pi * (x - 2));

    res.x = res.x == 0.0f ? 1.0f :
        __sinf(res.x) * __sinf(res.x / 2.0f) / (res.x * res.x / 2.0f);
    res.y = res.y == 0.0f ? 1.0f :
        __sinf(res.y) * __sinf(res.y / 2.0f) / (res.y * res.y / 2.0f);
    res.z = res.z == 0.0f ? 1.0f :
        __sinf(res.z) * __sinf(res.z / 2.0f) / (res.z * res.z / 2.0f);
    res.w = res.w == 0.0f ? 1.0f :
        __sinf(res.w) * __sinf(res.w / 2.0f) / (res.w * res.w / 2.0f);

    return res / (res.x + res.y + res.z + res.w);
}

__device__ static inline float4 bicubic_coeffs(float x, float param)
{
    const float A = param == SCALE_CUDA_PARAM_DEFAULT ? 0.0f : -param;

    float4 res;
    res.x = ((A * (x + 1) - 5 * A) * (x + 1) + 8 * A) * (x + 1) - 4 * A;
    res.y = ((A + 2) * x - (A + 3)) * x * x + 1;
    res.z = ((A + 2) * (1 - x) - (A + 3)) * (1 - x) * (1 - x) + 1;
    res.w = 1.0f - res.x - res.y - res.z;

    return res;
}

template<typename V>
__device__ static inline V apply_coeffs(float4 coeffs, V c0, V c1, V c2, V c3)
{
    V res = c0 * coeffs.x;
    res  += c1 * coeffs.y;
    res  += c2 * coeffs.z;
    res  += c3 * coeffs.w;

    return res;
}

template<typename T>
__device__ static inline T Subsample_Nearest(hipTextureObject_t tex,
                                             int xo, int yo,
                                             int dst_width, int dst_height,
                                             int src_width, int src_height,
                                             int bit_depth, float param)
{
    float hscale = (float)src_width / (float)dst_width;
    float vscale = (float)src_height / (float)dst_height;
    float xi = (xo + 0.5f) * hscale;
    float yi = (yo + 0.5f) * vscale;

    return tex2D<T>(tex, xi, yi);
}

template<typename T>
__device__ static inline T Subsample_Bilinear(hipTextureObject_t tex,
                                              int xo, int yo,
                                              int dst_width, int dst_height,
                                              int src_width, int src_height,
                                              int bit_depth, float param)
{
    float hscale = (float)src_width / (float)dst_width;
    float vscale = (float)src_height / (float)dst_height;
    float xi = (xo + 0.5f) * hscale;
    float yi = (yo + 0.5f) * vscale;
    // 3-tap filter weights are {wh,1.0,wh} and {wv,1.0,wv}
    float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
    float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
    // Convert weights to two bilinear weights -> {wh,1.0,wh} -> {wh,0.5,0} + {0,0.5,wh}
    float dx = wh / (0.5f + wh);
    float dy = wv / (0.5f + wv);

    intT r;
    vec_set_scalar(r, 2);
    r += tex2D<T>(tex, xi - dx, yi - dy);
    r += tex2D<T>(tex, xi + dx, yi - dy);
    r += tex2D<T>(tex, xi - dx, yi + dy);
    r += tex2D<T>(tex, xi + dx, yi + dy);

    T res;
    vec_set(res, r >> 2);

    return res;
}

template<typename T, coeffs_function_t coeffs_function>
__device__ static inline T Subsample_Bicubic(hipTextureObject_t tex,
                                             int xo, int yo,
                                             int dst_width, int dst_height,
                                             int src_width, int src_height,
                                             int bit_depth, float param)
{
    float hscale = (float)src_width / (float)dst_width;
    float vscale = (float)src_height / (float)dst_height;
    float xi = (xo + 0.5f) * hscale - 0.5f;
    float yi = (yo + 0.5f) * vscale - 0.5f;
    float px = floor(xi);
    float py = floor(yi);
    float fx = xi - px;
    float fy = yi - py;

    float factor = bit_depth > 8 ? 0xFFFF : 0xFF;

    float4 coeffsX = coeffs_function(fx, param);
    float4 coeffsY = coeffs_function(fy, param);

#define PIX(x, y) tex2D<floatT>(tex, (x), (y))

    return from_floatN<T, floatT>(
        apply_coeffs<floatT>(coeffsY,
            apply_coeffs<floatT>(coeffsX, PIX(px - 1, py - 1), PIX(px, py - 1), PIX(px + 1, py - 1), PIX(px + 2, py - 1)),
            apply_coeffs<floatT>(coeffsX, PIX(px - 1, py    ), PIX(px, py    ), PIX(px + 1, py    ), PIX(px + 2, py    )),
            apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 1), PIX(px, py + 1), PIX(px + 1, py + 1), PIX(px + 2, py + 1)),
            apply_coeffs<floatT>(coeffsX, PIX(px - 1, py + 2), PIX(px, py + 2), PIX(px + 1, py + 2), PIX(px + 2, py + 2))
        ) * factor
    );

#undef PIX
}

/// --- FUNCTION EXPORTS ---

#define KERNEL_ARGS(T) \
    hipTextureObject_t src_tex_0, hipTextureObject_t src_tex_1, \
    hipTextureObject_t src_tex_2, hipTextureObject_t src_tex_3, \
    T *dst_0, T *dst_1, T *dst_2, T *dst_3,                       \
    int dst_width, int dst_height, int dst_pitch,                 \
    int src_width, int src_height, float param

#define SUBSAMPLE(Convert, T) \
    hipTextureObject_t src_tex[4] =                    \
        { src_tex_0, src_tex_1, src_tex_2, src_tex_3 }; \
    T *dst[4] = { dst_0, dst_1, dst_2, dst_3 };         \
    int xo = blockIdx.x * blockDim.x + threadIdx.x;     \
    int yo = blockIdx.y * blockDim.y + threadIdx.y;     \
    if (yo >= dst_height || xo >= dst_width) return;    \
    Convert(                                            \
        src_tex, dst, xo, yo,                           \
        dst_width, dst_height, dst_pitch,               \
        src_width, src_height, param);

extern "C" {

#define NEAREST_KERNEL(C, S) \
    __global__ void Subsample_Nearest_##C##S(                      \
        KERNEL_ARGS(Convert_##C::out_T##S))                        \
    {                                                              \
        SUBSAMPLE((Convert_##C::Convert##S<                        \
                       Subsample_Nearest<Convert_##C::in_T>,       \
                       Subsample_Nearest<Convert_##C::in_T_uv> >), \
                  Convert_##C::out_T##S) \
    }

#define NEAREST_KERNEL_RAW(C) \
    NEAREST_KERNEL(C,)   \
    NEAREST_KERNEL(C,_uv)

#define NEAREST_KERNELS(C) \
    NEAREST_KERNEL_RAW(yuv420p_ ## C)     \
    NEAREST_KERNEL_RAW(nv12_ ## C)        \
    NEAREST_KERNEL_RAW(yuv444p_ ## C)     \
    NEAREST_KERNEL_RAW(p010le_ ## C)      \
    NEAREST_KERNEL_RAW(p016le_ ## C)      \
    NEAREST_KERNEL_RAW(yuv444p16le_ ## C)

NEAREST_KERNELS(yuv420p)
NEAREST_KERNELS(nv12)
NEAREST_KERNELS(yuv444p)
NEAREST_KERNELS(p010le)
NEAREST_KERNELS(p016le)
NEAREST_KERNELS(yuv444p16le)

NEAREST_KERNEL_RAW(bgr0_bgr0)
NEAREST_KERNEL_RAW(rgb0_rgb0)
NEAREST_KERNEL_RAW(bgr0_rgb0)
NEAREST_KERNEL_RAW(rgb0_bgr0)


#define BILINEAR_KERNEL(C, S) \
    __global__ void Subsample_Bilinear_##C##S(                      \
        KERNEL_ARGS(Convert_##C::out_T##S))                         \
    {                                                               \
        SUBSAMPLE((Convert_##C::Convert##S<                         \
                       Subsample_Bilinear<Convert_##C::in_T>,       \
                       Subsample_Bilinear<Convert_##C::in_T_uv> >), \
                  Convert_##C::out_T##S) \
    }

#define BILINEAR_KERNEL_RAW(C) \
    BILINEAR_KERNEL(C,)   \
    BILINEAR_KERNEL(C,_uv)

#define BILINEAR_KERNELS(C) \
    BILINEAR_KERNEL_RAW(yuv420p_ ## C)     \
    BILINEAR_KERNEL_RAW(nv12_ ## C)        \
    BILINEAR_KERNEL_RAW(yuv444p_ ## C)     \
    BILINEAR_KERNEL_RAW(p010le_ ## C)      \
    BILINEAR_KERNEL_RAW(p016le_ ## C)      \
    BILINEAR_KERNEL_RAW(yuv444p16le_ ## C)

BILINEAR_KERNELS(yuv420p)
BILINEAR_KERNELS(nv12)
BILINEAR_KERNELS(yuv444p)
BILINEAR_KERNELS(p010le)
BILINEAR_KERNELS(p016le)
BILINEAR_KERNELS(yuv444p16le)

BILINEAR_KERNEL_RAW(bgr0_bgr0)
BILINEAR_KERNEL_RAW(rgb0_rgb0)
BILINEAR_KERNEL_RAW(bgr0_rgb0)
BILINEAR_KERNEL_RAW(rgb0_bgr0)

#define BICUBIC_KERNEL(C, S) \
    __global__ void Subsample_Bicubic_##C##S(                                        \
        KERNEL_ARGS(Convert_##C::out_T##S))                                          \
    {                                                                                \
        SUBSAMPLE((Convert_##C::Convert##S<                                          \
                       Subsample_Bicubic<Convert_## C ::in_T, bicubic_coeffs>,       \
                       Subsample_Bicubic<Convert_## C ::in_T_uv, bicubic_coeffs> >), \
                  Convert_##C::out_T##S)                                             \
    }

#define BICUBIC_KERNEL_RAW(C) \
    BICUBIC_KERNEL(C,)   \
    BICUBIC_KERNEL(C,_uv)

#define BICUBIC_KERNELS(C) \
    BICUBIC_KERNEL_RAW(yuv420p_ ## C)     \
    BICUBIC_KERNEL_RAW(nv12_ ## C)        \
    BICUBIC_KERNEL_RAW(yuv444p_ ## C)     \
    BICUBIC_KERNEL_RAW(p010le_ ## C)      \
    BICUBIC_KERNEL_RAW(p016le_ ## C)      \
    BICUBIC_KERNEL_RAW(yuv444p16le_ ## C)

BICUBIC_KERNELS(yuv420p)
BICUBIC_KERNELS(nv12)
BICUBIC_KERNELS(yuv444p)
BICUBIC_KERNELS(p010le)
BICUBIC_KERNELS(p016le)
BICUBIC_KERNELS(yuv444p16le)

BICUBIC_KERNEL_RAW(bgr0_bgr0)
BICUBIC_KERNEL_RAW(rgb0_rgb0)
BICUBIC_KERNEL_RAW(bgr0_rgb0)
BICUBIC_KERNEL_RAW(rgb0_bgr0)


#define LANCZOS_KERNEL(C, S) \
    __global__ void Subsample_Lanczos_##C##S(                                        \
        KERNEL_ARGS(Convert_##C::out_T##S))                                          \
    {                                                                                \
        SUBSAMPLE((Convert_##C::Convert##S<                                          \
                       Subsample_Bicubic<Convert_## C ::in_T, lanczos_coeffs>,       \
                       Subsample_Bicubic<Convert_## C ::in_T_uv, lanczos_coeffs> >), \
                  Convert_##C::out_T##S) \
    }

#define LANCZOS_KERNEL_RAW(C) \
    LANCZOS_KERNEL(C,)   \
    LANCZOS_KERNEL(C,_uv)

#define LANCZOS_KERNELS(C) \
    LANCZOS_KERNEL_RAW(yuv420p_ ## C)     \
    LANCZOS_KERNEL_RAW(nv12_ ## C)        \
    LANCZOS_KERNEL_RAW(yuv444p_ ## C)     \
    LANCZOS_KERNEL_RAW(p010le_ ## C)      \
    LANCZOS_KERNEL_RAW(p016le_ ## C)      \
    LANCZOS_KERNEL_RAW(yuv444p16le_ ## C)

LANCZOS_KERNELS(yuv420p)
LANCZOS_KERNELS(nv12)
LANCZOS_KERNELS(yuv444p)
LANCZOS_KERNELS(p010le)
LANCZOS_KERNELS(p016le)
LANCZOS_KERNELS(yuv444p16le)

LANCZOS_KERNEL_RAW(bgr0_bgr0)
LANCZOS_KERNEL_RAW(rgb0_rgb0)
LANCZOS_KERNEL_RAW(bgr0_rgb0)
LANCZOS_KERNEL_RAW(rgb0_bgr0)

}
