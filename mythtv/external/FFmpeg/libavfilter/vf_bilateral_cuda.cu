#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022 Mohamed Khaled <Mohamed_Khaled_Kamal@outlook.com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include "cuda/vector_helpers.cuh"

extern "C"
{

/**
 * @brief               calculated squared norm difference between two 3-dimension vecors ||first_vector-second_vector||^2
 *                      used float4 for better performance
 *
 * @param first_yuv     first color vector
 * @param second_yuv    second color vecotr
 * @return              answer of squared norm difference
 */
__device__ static inline float norm_squared(float4 first_yuv, float4 second_yuv)
{
    float x = first_yuv.x - second_yuv.x;
    float y = first_yuv.y - second_yuv.y;
    float z = first_yuv.z - second_yuv.z;
    return (x*x) + (y*y) + (z*z);
}

/**
 * @brief               calculate w as stated in bilateral filter research paper
 *
 * @param first_yuv     first color vector
 * @param second_yuv    second color vecotr
 * @return              the calculated w
 */
__device__ static inline float calculate_w(int x, int y, int r, int c,
                                           float4 pixel_value, float4 neighbor_value,
                                           float sigma_space, float sigma_color)
{
    float first_term, second_term;
    first_term = (((x - r) * (x - r)) + ((y - c) * (y - c))) / (2 * sigma_space * sigma_space);
    second_term = norm_squared(pixel_value, neighbor_value) / (2 * sigma_color * sigma_color);
    return __expf(-first_term - second_term);
}

/**
 * @brief apply the bilateral filter on the pixel sent
 *
 * @param src_tex_Y         Y channel of source image
 * @param src_tex           U channel of source image if yuv, or UV channels if format is nv12
 * @param src_tex_V         V channel of source image
 * @param dst_Y             Y channel of destination image
 * @param dst_U             U channel of destination image if format is in yuv
 * @param dst_V             V channel of destination image if format is in yuv
 * @param dst_UV            UV channels of destination image if format is in nv12
 * @param width             width of Y channel
 * @param height            height of Y channel
 * @param width_uv          width of UV channels
 * @param height_uv         height of UV channels
 * @param pitch             pitch of Y channel
 * @param pitch_uv          pitch of UV channels
 * @param x                 x coordinate of pixel to be filtered
 * @param y                 y coordinate of pixel to be filtered
 * @param sigma_space       sigma space parameter
 * @param sigma_color       sigma color parameter
 * @param window_size       window size parameter
 * @return void
 */
__device__ static inline void apply_biltaeral(
    hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex, hipTextureObject_t src_tex_V,
    uchar *dst_Y, uchar *dst_U, uchar *dst_V, uchar2 *dst_UV,
    int width, int height, int width_uv, int height_uv, int pitch, int pitch_uv,
    int x, int y,
    float sigma_space, float sigma_color, int window_size)
{
    int start_r = x - window_size / 2;
    int start_c = y - window_size / 2;
    float4 neighbor_pixel = make_float4(0.f, 0.f, 0.f, 0.f);
    float Wp = 0.f;
    float4 new_pixel_value = make_float4(0.f, 0.f, 0.f, 0.f);
    float w = 0.f;

    int channel_ratio = width / width_uv; // ratio between Y channel and UV channels
    float4 currrent_pixel;

    if (!src_tex_V) { // format is in nv12
        float2 temp_uv   = tex2D<float2>(src_tex, x/channel_ratio, y/channel_ratio) * 255.f;
        currrent_pixel.x = tex2D<float>(src_tex_Y, x, y) * 255.f;
        currrent_pixel.y = temp_uv.x;
        currrent_pixel.z = temp_uv.y;
        currrent_pixel.w = 0.f;
    } else { // format is fully planar
        currrent_pixel = make_float4(tex2D<float>(src_tex_Y, x, y) * 255.f,
                                     tex2D<float>(src_tex,   x/channel_ratio, y/channel_ratio) * 255.f,
                                     tex2D<float>(src_tex_V, x/channel_ratio, y/channel_ratio) * 255.f,
                                     0.f);
    }

    for (int i=0; i < window_size; i++)
    {
        for (int j=0; j < window_size; j++)
        {
            int r=start_r+i;
            int c=start_c+j;
            bool in_bounds=r>=0 && r<width && c>=0 && c<height;
            if (in_bounds)
            {
                if (!src_tex_V){
                    float2 temp_uv = tex2D<float2>(src_tex, r/channel_ratio, c/channel_ratio);
                    neighbor_pixel=make_float4(tex2D<float>(src_tex_Y, r, c) * 255.f,
                                               temp_uv.x * 255.f,
                                               temp_uv.y * 255.f, 0.f);
                } else {
                    neighbor_pixel=make_float4(tex2D<float>(src_tex_Y, r, c) * 255.f,
                                               tex2D<float>(src_tex, r/channel_ratio, c/channel_ratio) * 255.f,
                                               tex2D<float>(src_tex_V, r/channel_ratio, c/channel_ratio) * 255.f, 0.f);
                }
                w=calculate_w(x,y,r,c,currrent_pixel,neighbor_pixel,sigma_space,sigma_color);
                Wp+=w;
                new_pixel_value+= neighbor_pixel*w;
            }
        }
    }

    new_pixel_value    = new_pixel_value / Wp;
    dst_Y[y*pitch + x] = new_pixel_value.x;

    if (!src_tex_V) {
        dst_UV[(y/channel_ratio) * pitch_uv + (x/channel_ratio)] = make_uchar2(new_pixel_value.y, new_pixel_value.z);
    } else {
        dst_U[(y/channel_ratio) * pitch_uv + (x/channel_ratio)]  = new_pixel_value.y;
        dst_V[(y/channel_ratio) * pitch_uv + (x/channel_ratio)]  = new_pixel_value.z;
    }

    return;
}


__global__ void Process_uchar(hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_U, hipTextureObject_t src_tex_V,
                              uchar *dst_Y, uchar *dst_U, uchar *dst_V,
                              int width, int height, int pitch,
                              int width_uv, int height_uv, int pitch_uv,
                              int window_size, float sigmaS, float sigmaR)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width)
        return;

    apply_biltaeral(src_tex_Y, src_tex_U, src_tex_V,
                    dst_Y, dst_U, dst_V, (uchar2*)nullptr,
                    width, height, width_uv, height_uv, pitch, pitch_uv,
                    x, y,
                    sigmaS, sigmaR, window_size);
}


__global__ void Process_uchar2(hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_UV, hipTextureObject_t unused1,
                               uchar *dst_Y, uchar2 *dst_UV, uchar *unused2,
                               int width, int height, int pitch,
                               int width_uv, int height_uv, int pitch_uv,
                               int window_size, float sigmaS, float sigmaR)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width)
        return;

    apply_biltaeral(src_tex_Y, src_tex_UV, (hipTextureObject_t)nullptr,
                    dst_Y, (uchar*)nullptr, (uchar*)nullptr, dst_UV,
                    width, height, width_uv, height_uv, pitch, pitch_uv,
                    x, y,
                    sigmaS, sigmaR, window_size);
}

}
